#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "hip/device_functions.h"

#include <iostream>
#include <stdio.h>
#include <ctime>

void random_longs(long* a, int size) //randomizing elements in both vectors
{
	srand(time(0));
	int i;
	for (i = 0; i < size; ++i)
	{
		a[i] = rand() % 100;
	}
}

__global__ void add(long* a, long* b, long* c, long N) { //core from ScalarMultiplication_example1
	long baseIdx = threadIdx.x;
	long idx = baseIdx;
	while (idx < N)
	{
		c[idx] = a[idx] * b[idx];
		idx += blockDim.x;
	}
	__syncthreads();
	long step = N / 2;
	while (step != 0) {
		idx = baseIdx;
		while (idx < step) {
			c[idx] += c[idx + step];
			idx += blockDim.x;
		}
		step /= 2;
		__syncthreads();
	}
}

__global__ void add_1024(long* a, long* b, long* c, long N) { //more simple and probably faster core but works only with 1024 or less elements in vector in this example
	c[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];
	__syncthreads();
	long step = N / 2;
	while (step != 0) {
		if (threadIdx.x < step)
		{
			c[threadIdx.x] += c[threadIdx.x + step];
		}
		step /= 2;
		__syncthreads();
	}
}

int main(void) {
	long N;
	clock_t start_t_gpu, start_t_cpu, end_t_gpu, end_t_cpu;
	long res_CPU;
	long* a, * b, * c, * d; // host copies of a, b, c
	long* d_a, * d_b, * d_c; // device copies of a, b, c
	int size;
	for (int i = 0; i < 3; i++)
	{
		if (i == 0) N = 512;
		if (i == 1) N = 1024;
		if (i == 2) N = 131072;
		size = N * sizeof(long);
		std::cout << "Vector size: " << N << std::endl;
		// Alloc space for device copies of a, b, c
		hipMalloc((void**)& d_a, size);
		hipMalloc((void**)& d_b, size);
		hipMalloc((void**)& d_c, size);
		// Alloc space for host copies of a, b, c and setup input values
		a = (long*)malloc(size); random_longs(a, N);
		b = (long*)malloc(size); random_longs(b, N);
		c = (long*)malloc(sizeof(long));
		d = (long*)malloc(size);
		// Copy inputs to device
		hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
		hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
		// Launch add() kernel on GPU with N blocks

		if (N <= 1024)
		{
			start_t_gpu = clock();
			for (int i = 0; i < 10000; i++)
			{
				add_1024 << <1, N >> > (d_a, d_b, d_c, N);
			}
			end_t_gpu = clock();
			hipMemcpy(c, d_c, sizeof(long), hipMemcpyDeviceToHost);
			// Copy result back to host
			std::cout << "ScalarMultiplication on GPU (simple core): " << std::endl << "result: " << c[0] << std::endl;
			std::cout << "time: " << ((double)end_t_gpu - start_t_gpu)/ CLOCKS_PER_SEC << " seconds" << std::endl;
		}
		else {
			start_t_gpu = clock();
			for (int i = 0; i < 10000; i++)
			{
				add << <1, 1024 >> > (d_a, d_b, d_c, N);
			}
			hipMemcpy(c, d_c, sizeof(long), hipMemcpyDeviceToHost);
			end_t_gpu = clock();
			// Copy result back to host
			std::cout << "ScalarMultiplication on GPU (core from ScalarMultiplication_example1): " << std::endl << "result: " << c[0] << std::endl;
			std::cout << "time: " << ((double)end_t_gpu - start_t_gpu)/ CLOCKS_PER_SEC << " seconds" << std::endl;
		}

		start_t_cpu = clock();
		for (int i = 0; i < 10000; i++) //cycle just to see average time
		{
			res_CPU = 0;
			for (long i = 0; i < N; ++i)
			{
				res_CPU += a[i] * b[i];
			}
		}
		end_t_cpu = clock();
		std::cout << "ScalarMultiplication on CPU: " << std::endl << "result: " << res_CPU << std::endl;
		std::cout << "time: " << ((double)end_t_cpu - start_t_cpu)/ CLOCKS_PER_SEC << " seconds" << std::endl << std::endl;
		// Cleanup
		free(a); free(b); free(c);
		hipFree(d_a); hipFree(d_b); hipFree(d_c);
	}
	return 0;
}